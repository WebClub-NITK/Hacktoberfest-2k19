#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
using namespace std;

__global__
void MaxSizeSubMat(int* d_Matrix, int* d_Soln, int* d_clmns, int* d_rows, int* N) {
	int threadId = threadIdx.x, i , j;
	if(*N >= (*d_clmns + *d_rows)/2) {
		i = *d_rows - (threadId + 1);
		j = *N + threadId + 2 - *d_rows;
	} else {
		i = *N - threadId;
		j = threadId + 1;
	}
	if(j >= *d_clmns || j <= 0 || i <= 0 || i >= *d_rows) {
		return;
	}
	if(d_Matrix[ (*d_clmns * i) + j ] == 1) {

		int a = d_Soln[ (*d_clmns * i) + j-1];
		int b = d_Soln[ (*d_clmns * (i-1)) + j ];
		int c = d_Soln[ (*d_clmns * (i-1)) + j -1 ];

		if(a <= b && a <= c) {
			d_Soln[ (*d_clmns * i) + j ] = a + 1;
		} else if(b <= c && b <= a) {
			d_Soln[ (*d_clmns * i) + j ] = b + 1;
		}else {
			d_Soln[ (*d_clmns * i) + j ] = c + 1;
		}

	} else {
		d_Soln[ (*d_clmns * i) + j ] = 0;
	}
	
	__syncthreads();
	if(threadId == 0) {
		*N += 1;
	}
}
int main() {
	int intsize = sizeof(int) ;
	int h_rows, h_clmns;

	cout << "Enter the Number of rows in the matrix." << endl;
	cin >> h_rows;
	cout << "Enter the Number of columns in the matrix." << endl;
	cin >> h_clmns;

	int h_Matrx[h_rows][h_clmns];

	int  h_Soln[h_rows][h_clmns];
	cout << "Enter the elements of the matrix.\n";
	for(int i = 0; i < h_rows; i++) {
		for(int j = 0; j < h_clmns; j++) {
			cin >> h_Matrx[i][j];
			if(i == 0 || j == 0) {
				h_Soln[i][j] = h_Matrx[i][j];
			} else {
				h_Soln[i][j] = 0;
			}
		}
	}

	int *d_Matrix, *d_clmns, *d_Soln, *d_rows;

	hipMalloc((void**)&d_Matrix, h_rows * h_clmns * intsize);
	hipMalloc((void**)&d_Soln, h_rows * h_clmns * intsize);
	hipMalloc((void**)&d_clmns, intsize);
	hipMalloc((void**)&d_rows, intsize);

	hipMemcpy( d_Matrix, h_Matrx, h_rows * h_clmns * intsize, hipMemcpyHostToDevice);
	hipMemcpy( d_Soln, h_Soln, h_rows * h_clmns * intsize, hipMemcpyHostToDevice);
	hipMemcpy(d_clmns, &h_clmns, intsize, hipMemcpyHostToDevice);
	hipMemcpy(d_rows, &h_rows, intsize, hipMemcpyHostToDevice);

	int n = (h_rows + h_clmns - 3), *N;

	hipMalloc((void**)&N, intsize);

	int i = 1;

	hipMemcpy( N, &i, intsize, hipMemcpyHostToDevice);

	for(int i = 1; i <= (n + 1)/2; i++) {
		MaxSizeSubMat<<<1,i>>> (d_Matrix, d_Soln, d_clmns, d_rows, N);
	}

	for(int i = (n + 3)/2; i <= n ; i++ ) {
		MaxSizeSubMat<<<1, (n - i + 1)>>> (d_Matrix, d_Soln, d_clmns, d_rows, N);
	}
	
	hipMemcpy( h_Soln, d_Soln, h_rows * h_clmns * intsize, hipMemcpyDeviceToHost);

	int max = h_Soln[0][0];

	for(int i = 0; i < h_rows; i++) {
		for(int j = 0; j < h_clmns; j++) {
			if(max < h_Soln[i][j]) {
				max = h_Soln[i][j];
			}
	}
	cout << "Max area is " << max * max << ".\n";

	hipFree(d_Matrix); hipFree(d_Soln);
}

